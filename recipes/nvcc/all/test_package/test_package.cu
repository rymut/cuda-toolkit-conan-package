
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(int *a, int *b) {
    a[threadIdx.x] = b[threadIdx.x];
}
int main(void)
{
    int devs = 0;
    hipError_t status = hipGetDeviceCount(&devs);
    printf("devs available: %d (status %d: \"%s\")\n", devs, status, hipGetErrorString(status));
    return EXIT_SUCCESS;
}
